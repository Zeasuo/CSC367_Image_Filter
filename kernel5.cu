#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include <hip/hip_runtime.h>
#include "kernels.h"
#include <stdio.h>
#include <string>
#include <unistd.h>
#include <sys/sysinfo.h>
#include <time.h>
#include <stdint.h>
#define max_threads_block 1024
#define min(x ,y) ((x<y)?x:y)
#define max(x ,y) ((x>y)?x:y)

__constant__ int8_t log_filter[] = {
    0, 1, 1, 2, 2, 2,   1,   1,   0, 1, 2, 4, 5, 5,   5,   4,   2,
    1, 1, 4, 5, 3, 0,   3,   5,   4, 1, 2, 5, 3, -12, -24, -12, 3,
    5, 2, 2, 5, 0, -24, -40, -24, 0, 5, 2, 2, 5, 3,   -12, -24, -12,
    3, 5, 2, 1, 4, 5,   3,   0,   3, 5, 4, 1, 1, 2,   4,   5,   5,
    5, 4, 2, 1, 0, 1,   1,   2,   2, 2, 1, 1, 0,
};

void run_kernel5(int32_t dimension, const int32_t *input, int32_t *output, int32_t width, int32_t height) {
    // Figure out how to split the work into threads and call the kernel below.
    int nPixel = width * height;
    int32_t threads_needed = min(max_threads_block, nPixel);
    int32_t block_needed = (nPixel + threads_needed -1)/threads_needed;
    int32_t reduction_blocks;
    int32_t reduction_threads;
    int32_t iteration_n = nPixel;
    dim3 threads(threads_needed, 1);
    dim3 blocks(block_needed);
    kernel5 <<<blocks, threads>>> (dimension, input, output, width, height);
    
    int32_t *global_mins;
    int32_t *global_maxs;
    hipMalloc(&global_mins, width*height*sizeof(int32_t));
    hipMalloc(&global_maxs, width*height*sizeof(int32_t));
    bool should_repeat = my_calculate_blocks_and_threads(iteration_n, reduction_blocks, reduction_threads);
    gpu_switch_threads(iteration_n, reduction_threads, reduction_blocks, output, global_mins, global_maxs, 1);
    while(should_repeat){
        iteration_n = reduction_blocks;
        should_repeat = my_calculate_blocks_and_threads(iteration_n, reduction_blocks, reduction_threads);
        gpu_switch_threads(iteration_n, reduction_threads, reduction_blocks, output, global_mins, global_maxs, 0);//the output parameter passed here does not matter as it will not be used in the function
    }

    normalize5 <<<blocks, threads>>> (output, width, height, global_mins, global_maxs);
    hipFree(global_mins);
    hipFree(global_maxs);
}

__global__ void kernel5(int32_t dimension, const int32_t *input, int32_t *output, int32_t width, int32_t height) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int32_t row = idx / width;
    int32_t col = idx % width;
    int index = (row*width)+col;
    if (index < width*height && idx < width*height){
        output[index] = apply2dGPU(log_filter, dimension, input, width, height, row, col);
    }
}

__global__ void normalize5(int32_t *image, int32_t width, int32_t height, int32_t *smallest, int32_t *biggest) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int32_t row = idx / width;
    int32_t col = idx % width;
    int index = row*width+col;
    if (index < height * width){
        if (smallest[0] == biggest[0]) {
            return;
        }
        image[index] = ((image[index] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
    }
}
