#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include <hip/hip_runtime.h>
#include "kernels.h"
#include <stdio.h>
#include <string>
#include <unistd.h>
#include <sys/sysinfo.h>
#include <time.h>
#include <stdint.h>
#define max_threads_block 1024
#define max_blocks 36 //This is the max number of block can be used

void run_kernel4(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height) {
  // Figure out how to split the work into threads and call the kernel below.
  int nPixel = width * height;
  int32_t threads_needed = min(max_threads_block, nPixel);
  int32_t block_needed = min(max_blocks, (nPixel + threads_needed - 1) / threads_needed);
  int32_t reduction_blocks;
  int32_t reduction_threads;
  int32_t iteration_n = nPixel;
  dim3 threads(threads_needed, 1);
  dim3 blocks(block_needed);

  kernel4 <<<blocks, threads>>> (filter, dimension, input, output, width, height);
  int32_t *global_mins;
  int32_t *global_maxs;
  hipMalloc(&global_mins, width*height*sizeof(int32_t));
  hipMalloc(&global_maxs, width*height*sizeof(int32_t));
  bool should_repeat = my_calculate_blocks_and_threads(iteration_n, reduction_blocks, reduction_threads);
  gpu_switch_threads(iteration_n, reduction_threads, reduction_blocks, output, global_mins, global_maxs, 1);

  while(should_repeat){
      iteration_n = reduction_blocks;
      should_repeat = my_calculate_blocks_and_threads(iteration_n, reduction_blocks, reduction_threads);
      gpu_switch_threads(iteration_n, reduction_threads, reduction_blocks, output, global_mins, global_maxs, 0);//the output parameter passed here does not matter as it will not be used in the function
  }


  normalize4 <<<blocks, threads>>> (output, width, height, global_mins, global_maxs);
  hipFree(global_mins);
  hipFree(global_maxs);
}

__global__ void kernel4(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height) {
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = idx; i<width*height;i+=stride){
    int row = i/width;
    int col = i % width;
    output[i] = apply2dGPU(filter, dimension, input, width, height, row, col);
  }
}

__global__ void normalize4(int32_t *image, int32_t width, int32_t height,
                           int32_t *smallest, int32_t *biggest) {
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = idx; i<width*height;i+=stride){
    if (smallest[0] != biggest[0]) {
      		image[i] = ((image[i] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
    }
  }
}
